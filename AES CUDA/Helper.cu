/*********************************************************************/
/*                       INCLUDES AND DEFINES                        */
/*********************************************************************/

#include <vector>
#include <iostream>

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>

#include "Helper.h"

using std::cout;
using std::endl;
using std::vector;

/*********************************************************************/
/*                          HELPER FUNCTIONS                         */
/*********************************************************************/

// Cout whole ByteArray
void print_byte_array(unsigned char *arr)
{
	for (size_t i = 0; i != sizeof(arr) / sizeof(arr[0]); ++i)
	{
		cout << std::hex << (int)arr[i] << "\t";
	}
	cout << endl << endl;
}


// Cout hex byte
void print_byte(const unsigned char &byte)
{
	cout << endl << "Byte: " << std::hex << (int)byte;
}