/*********************************************************************/
/*                       INCLUDES AND DEFINES                        */
/*********************************************************************/

#include <vector>
#include <iostream>

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>

#include "Helper.h"

using std::cout;
using std::endl;
using std::vector;

/*********************************************************************/
/*                          HELPER FUNCTIONS                         */
/*********************************************************************/

// Cout whole ByteArray
void print_byte_array(ByteArray &arr)
{
	for (size_t i = 0; i != arr.size(); ++i)
	{
		cout << std::hex << (int)arr[i] << "\t";
	}
	cout << endl << endl;
}


// Cout hex byte
void print_byte(const unsigned char &byte)
{
	cout << endl << "Byte: " << std::hex << (int)byte;
}