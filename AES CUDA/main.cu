#include "hip/hip_runtime.h"
/*********************************************************************/
/*                       INCLUDES AND DEFINES                        */
/*********************************************************************/

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>

#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <vector>
#include <chrono>
#include <string>

#include "Helper.h"
#include "AES.h"
#include "timer.h"

using std::cout;
using std::endl;
using std::vector;
using std::string;

#define THREADS_PER_BLOCK 128
#define ROUNDS 10

/*********************************************************************/
/*                       GPU HELPER FUNCTIONS                        */
/*********************************************************************/

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		getchar();
		if (abort) exit(code);
	}
}

/*********************************************************************/
/*                       COUNTER MODE FUNCTIONS                      */
/*********************************************************************/

void counter_launch_kernel(unsigned char *messages, unsigned char *results, unsigned char *keys,
							const unsigned int &message_size, const unsigned int &filesize)
{
	float milliseconds = 0.0f;

	// Define launch config
	int chunks = filesize / KEY_BLOCK;
	int ThreadsPerBlock = THREADS_PER_BLOCK;
	int Blocks = ceil(chunks / ThreadsPerBlock);

	// Results to device memory
	unsigned char *d_results;
	gpuErrchk(hipMalloc((void **)&d_results, message_size * sizeof(unsigned char)));
	gpuErrchk(hipMemcpy(d_results, messages, message_size * sizeof(unsigned char), hipMemcpyHostToDevice));

	// SBOX to device memory
	unsigned char *d_sbox;
	gpuErrchk(hipMalloc((void **)&d_sbox, 256 * sizeof(unsigned char)));
	gpuErrchk(hipMemcpy(d_sbox, h_sbox, 256 * sizeof(unsigned char), hipMemcpyHostToDevice));

	// Subkeys to device memory
	unsigned char *d_keys;
	gpuErrchk(hipMalloc((void **)&d_keys, NUM_ROUNDS * KEY_BLOCK * sizeof(unsigned char)));
	gpuErrchk(hipMemcpy(d_keys, keys, NUM_ROUNDS * KEY_BLOCK * sizeof(unsigned char), hipMemcpyHostToDevice));

	for (int i = 0; i != ROUNDS; ++i)
	{
		GpuTimer timer;
		timer.Start();
		aes_encryption << <Blocks, ThreadsPerBlock >> > (d_sbox, d_results, d_keys, message_size);
		hipDeviceSynchronize();
		hipDeviceSynchronize();
		timer.Stop();
		milliseconds += timer.ElapsedMilliSeconds();
	}

	cout << "Done Counter Mode in: " << milliseconds / (float) NUM_ROUNDS << " (ms)." << endl;

	gpuErrchk(hipMemcpy(results, d_results, message_size * sizeof(unsigned char), hipMemcpyDeviceToHost));
	gpuErrchk(hipFree(d_results));
}

/*********************************************************************/
/*                        MAIN FUNCTION CALL                         */
/*********************************************************************/

int main()
{
	for (int i = 6; i > 5; i--)
	{
		cout << endl << "Text" << i;
		string file_path_key = "C:/Users/Jan/Dropbox/Master AI/Parallel Computing/Project/key.txt";
		string file_path_messages = "C:/Users/Jan/Dropbox/Master AI/Parallel Computing/Project/text" + std::to_string(i) + ".txt";
		int filesize = file_size(file_path_messages.c_str());

		// Load data from files
		unsigned char *key = read_key(file_path_key);
		unsigned char *keys = key_schedule(key);

		cout << endl << std::dec << "Starting AES CUDA - COUNTER MODE" << endl;

		// Read in data
		unsigned char * plaintexts = (unsigned char *)malloc(sizeof(unsigned char)*filesize);
		read_datafile(file_path_messages.c_str(), plaintexts);

		// Malloc Memory for Enc/Decrypted Solutions
		unsigned char *decrypted_solution;
		unsigned char *encrypted_solution;

		decrypted_solution = new unsigned char[filesize];
		encrypted_solution = new unsigned char[filesize];

		cout << endl << "Ready to start!" << endl << endl;

		// Starting Encryption
		cout << endl << "Starting AES CUDA - COUNTER MODE KERNEL " << endl;
		counter_launch_kernel(plaintexts, encrypted_solution, keys, filesize, filesize);

		// Starting Decryption
		cout << endl << "Starting AES CUDA - INVERSE COUNTER MODE KERNEL " << endl;
		counter_launch_kernel(encrypted_solution, decrypted_solution, keys, filesize, filesize);

		cout << "Plain: " << endl;

		for (int i = 0; i != 16; ++i)
		{
			cout << std::dec << (int)plaintexts[i];
		}

		cout << "Enc: " << endl;

		for (int i = 0; i != 16; ++i)
		{
			cout << std::dec << (int) encrypted_solution[i];
		}

		cout << "Dec: " << endl;

		for (int i = 0; i != 16; ++i)
		{
			cout << std::dec << (int)decrypted_solution[i];
		}

		// Checking if Decryption of Encryption is the plaintext
		cout << endl << "Legit solution: " << check_byte_arrays(plaintexts, decrypted_solution, filesize) << endl;
	}

	getchar();
}