/*********************************************************************/
/*                       INCLUDES AND DEFINES                        */
/*********************************************************************/

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>

#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <vector>
#include <chrono>
#include <string>
#include <tuple>

#include "Helper.h"
#include "AES.h"
#include "timer.h"

using std::cout;
using std::endl;
using std::vector;
using std::string;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

int file_size(const std::string &add) 
{
	ifstream mySource;
	mySource.open(add, std::ios_base::binary);
	mySource.seekg(0, std::ios_base::end);
	int size = mySource.tellg();
	mySource.close();
	return size;
}

/*********************************************************************/
/*                       COUNTER MODE FUNCTIONS                      */
/*********************************************************************/

void counter_launch_kernel(unsigned char **messages, unsigned char **results, 
							unsigned char *key, const unsigned int &message_size, 
							const unsigned int &filesize)
{
	float milliseconds = 0.0f;

	// Define launch config
	int ThreadsPerBlock = 1024;
	int Blocks = (filesize / 16) / ThreadsPerBlock;

	// Push subkeys to device memory
	unsigned char *keys = key_schedule(key);
	unsigned char *d_keys;
	const int size_keys = NUM_ROUNDS * KEY_BLOCK * sizeof(unsigned char);
	d_keys = new unsigned char[size_keys];
	gpuErrchk(hipMalloc((void **)&d_keys, size_keys));
	gpuErrchk(hipMemcpy(d_keys, keys, size_keys, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_keySchedule), &d_keys, sizeof(size_keys)));

	// Pushes results to device memory
	unsigned char *d_results;
	const int size_results = KEY_BLOCK * sizeof(unsigned char);
	d_results = new unsigned char[size_results];
	gpuErrchk(hipMalloc((void **)&d_results, size_results));
	gpuErrchk(hipMemcpy(d_results, messages, size_results, hipMemcpyHostToDevice));

	GpuTimer timer;
	timer.Start();
	cuda_aes_encrypt_ctr << <Blocks, ThreadsPerBlock >> > (d_results);
	hipDeviceSynchronize();
	hipDeviceSynchronize();
	timer.Stop();
	milliseconds = timer.ElapsedMilliSeconds();
	cout << "Done Counter Mode in: " << milliseconds << " (ms)." << endl;

	hipMemcpy(results, d_results, size_results, hipMemcpyDeviceToHost);
	hipFree(d_results);
}

/*********************************************************************/
/*                        MAIN FUNCTION CALL                         */
/*********************************************************************/

int main()
{
	string file_path_key = "C:/Users/Jan/Dropbox/Master AI/Parallel Computing/Project/key.txt";
	string file_path_messages = "C:/Users/Jan/Dropbox/Master AI/Parallel Computing/Project/test.txt";
	int filesize = file_size(file_path_messages);

	// Load data from files
	unsigned char *key = read_key(file_path_key);

	cout << endl << "Starting AES CUDA - COUNTER MODE, with Key: " << endl;
	print_byte_array(key);

	std::tuple<unsigned char**, size_t> t = read_datafile(file_path_messages);
	unsigned char **messages = std::get<0>(t);
	size_t message_size = std::get<1>(t) * KEY_BLOCK;

	// Malloc Memory for Enc/Decrypted Solutions
	unsigned char **decrypted_solution;
	unsigned char **encrypted_solution;

	decrypted_solution = new unsigned char*[message_size];
	encrypted_solution = new unsigned char*[message_size];

	for (int i = 0; i != message_size; ++i)
	{
		decrypted_solution[i] = 0x00;
		encrypted_solution[i] = 0x00;
	}

	// Starting Encryption
	cout << endl << "Starting AES CUDA - COUNTER MODE KERNEL " << endl;
	counter_launch_kernel(messages, encrypted_solution, key, message_size, filesize);

	// Starting Decryption
	cout << endl << "Starting AES CUDA - INVERSE COUNTER MODE KERNEL " << endl;
	counter_launch_kernel(encrypted_solution, decrypted_solution, key, message_size, filesize);

	cout << endl << "Legit solution: " << check_byte_arrays(messages, decrypted_solution, message_size * KEY_BLOCK) << endl;

	getchar();
}