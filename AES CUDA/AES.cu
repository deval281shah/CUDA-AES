#include "hip/hip_runtime.h"
/*********************************************************************/
/*                       INCLUDES AND DEFINES                        */
/*********************************************************************/

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>

#include <iostream>
#include <stdlib.h>
#include <vector>

#include "AES.h"
#include "Helper.h"

using std::cout;
using std::endl;
using std::vector;

/*********************************************************************/
/*                        SUB LAYER KERNEL                           */
/*********************************************************************/

// Byte substitution (S-Boxes) can be parallel
__global__ void byte_sub_kernel(unsigned char *message)
{
	int id = blockIdx.x*blockDim.x + threadIdx.x;

	if (id < KEY_BLOCK)
	{
		message[id] = d_sbox[message[id]];
	}
}

// Inverse byte substitution (S-Boxes) can be parallel
__global__ void byte_sub_inv_kernel(unsigned char *message)
{
	int id = blockIdx.x*blockDim.x + threadIdx.x;

	if (id < KEY_BLOCK)
	{
		message[id] = d_sboxinv[message[id]];
	}
}

// Shift rows - can be parallel
// B0, B4, B8, B12 stays the same
__global__ void shift_rows_kernel(unsigned char *message)
{
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned char j = 0, k = 0;

	if (id < SHIFT_ROW_LIMIT)
	{
		if (id == 0)
		{
			j = message[1];
			message[1] = message[5];
			message[5] = message[9];
			message[9] = message[13];
			message[13] = j;
		}
		else if (id == 1)
		{
			j = message[10];
			k = message[14];
			message[10] = message[2];
			message[2] = j;
			message[14] = message[6];
			message[6] = k;
		}
		else
		{
			k = message[3];
			message[3] = message[15];
			message[15] = message[11];
			message[11] = message[7];
			message[7] = k;
		}
	}
}

// Inverse shift rows - can be parallel
// C0, C4, C8, C12 stays the same
__global__ void shift_rows_inv_kernel(unsigned char *message)
{
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned char j = 0, k = 0;

	if (id < SHIFT_ROW_LIMIT)
	{
		if (id == 0)
		{
			j = message[1];
			message[1] = message[13];
			message[13] = message[9];
			message[9] = message[5];
			message[5] = j;
		}
		else if (id == 1)
		{
			j = message[2];
			k = message[6];
			message[2] = message[10];
			message[10] = j;
			message[6] = message[14];
			message[14] = k;
		}
		else
		{
			j = message[3];
			message[3] = message[7];
			message[7] = message[11];
			message[11] = message[15];
			message[15] = j;
		}
	}
}

// Mix column - can be parallel
__global__ void mix_columns_kernel(unsigned char *message)
{
	unsigned char b0, b1, b2, b3;
	int id = blockIdx.x*blockDim.x + threadIdx.x;

	if (id < MIX_COLUMN_LIMIT)
	{
		b0 = message[id + 0];
		b1 = message[id + 1];
		b2 = message[id + 2];
		b3 = message[id + 3];

		// Mix-Col Matrix * b vector
		message[id + 0] = d_mul[b0][0] ^ d_mul[b1][1] ^ b2 ^ b3;
		message[id + 1] = b0 ^ d_mul[b1][0] ^ d_mul[b2][1] ^ b3;
		message[id + 2] = b0 ^ b1 ^ d_mul[b2][0] ^ d_mul[b3][1];
		message[id + 3] = d_mul[b0][1] ^ b1 ^ b2 ^ d_mul[b3][0];
	}
}

// Inverse mix column
__global__ void mix_columns_inv_kernel(unsigned char *message)
{
	unsigned char c0, c1, c2, c3;
	int id = blockIdx.x*blockDim.x + threadIdx.x;

	if (id < MIX_COLUMN_LIMIT)
	{
		c0 = message[id + 0];
		c1 = message[id + 1];
		c2 = message[id + 2];
		c3 = message[id + 3];

		// Mix-Col Inverse Matrix * c vector
		message[id + 0] = d_mul[c0][5] ^ d_mul[c1][3] ^ d_mul[c2][4] ^ d_mul[c3][2];
		message[id + 1] = d_mul[c0][2] ^ d_mul[c1][5] ^ d_mul[c2][3] ^ d_mul[c3][4];
		message[id + 2] = d_mul[c0][4] ^ d_mul[c1][2] ^ d_mul[c2][5] ^ d_mul[c3][3];
		message[id + 3] = d_mul[c0][3] ^ d_mul[c1][4] ^ d_mul[c2][2] ^ d_mul[c3][5];
	}
}

// Key Addition Kernel
__global__ void key_addition_kernel(unsigned char *message, unsigned char *subkey)
{
	int id = blockIdx.x*blockDim.x + threadIdx.x;

	if (id < KEY_BLOCK)
	{
		message[id] = message[id] ^ subkey[id];
	}
}

/*********************************************************************/
/*                           CONSTRUCTORS                            */
/*********************************************************************/

// Constructor of AES en/decryption
AES::AES(unsigned char *key) : m_subkeys(SUB_KEYS)
{
	m_key = key;
	key_schedule();
}

/*********************************************************************/
/*                       EN- DECRYPTION FUNCTIONS                    */
/*********************************************************************/

// Starting the encryption phase
unsigned char* AES::encrypt(unsigned char *message)
{
	register int round = 0;

	// Key-Add before round 1 (R0)
	key_addition(message, round);
	round = 1;

	// Round 1 to NUM_ROUNDS - 1 (R1 to R9)
	for (round; round != NUM_ROUNDS; round++)
	{
		byte_sub(message);
		shift_rows(message);
		mix_columns(message);
		key_addition(message, round);
	}

	// Last round without Mix-Column (RNUM_ROUNDS)
	round = NUM_ROUNDS;
	byte_sub(message);
	shift_rows(message);
	key_addition(message, round);

	return message;
}

// Starting the decryption phase
unsigned char* AES::decrypt(unsigned char *message)
{
	register int round = NUM_ROUNDS;

	// Key-Add before round (Inverse NUM_ROUNDS)
	key_addition(message, round);
	shift_rows_inv(message);
	byte_sub_inv(message);
	round = NUM_ROUNDS - 1;

	// Round NUM_ROUNDS - 1 to 1 (Inverse R9 to R1)
	for (round; round > 0; round--)
	{
		key_addition(message, round);
		mix_columns_inv(message);
		shift_rows_inv(message);
		byte_sub_inv(message);
	}

	// Last round without Mix-Column (Inverse R0)
	round = 0;
	key_addition(message, round);

	return message;
}

/*********************************************************************/
/*                           KEY FUNCTIONS                           */
/*********************************************************************/

// Computing the round keys
void AES::key_schedule()
{
	register int r;

	for (r = 0; r != SUB_KEYS; r++)
	{
		if (r == 0)
			m_subkeys[r] = m_key;
		else
		{
			if (AES_BITS == 128)
				m_subkeys[r] = sub_key128(m_subkeys[r - 1], r - 1);
			else
				cout << "TODO! 192-bit and 256-bit not implemented yet." << endl;
		}
	}
}

// Computing subkeys for round 1 up to 10
unsigned char* AES::sub_key128(unsigned char *prev_subkey, const int &r)
{
	unsigned char *result;
	result = new unsigned char[KEY_BLOCK];

	register int i;

	result[0] = (prev_subkey[0] ^ (sbox[prev_subkey[13]] ^ RC[r]));
	result[1] = (prev_subkey[1] ^ sbox[prev_subkey[14]]);
	result[2] = (prev_subkey[2] ^ sbox[prev_subkey[15]]);
	result[3] = (prev_subkey[3] ^ sbox[prev_subkey[12]]);

	for (i = 4; i != KEY_BLOCK; i += 4)
	{
		result[i + 0] = (result[i - 4] ^ prev_subkey[i + 0]);
		result[i + 1] = (result[i - 3] ^ prev_subkey[i + 1]);
		result[i + 2] = (result[i - 2] ^ prev_subkey[i + 2]);
		result[i + 3] = (result[i - 1] ^ prev_subkey[i + 3]);
	}

	return result;
}

/*********************************************************************/
/*                              SUB LAYER                            */
/*********************************************************************/

// Byte substitution (S-Boxes) can be parallel
void AES::byte_sub(unsigned char *message)
{
	dim3 dim_block(KEY_BLOCK);
	dim3 dim_grid(1);

	int size_message = sizeof(message);
	unsigned char *d_message;
	hipMalloc((void **)&d_message, size_message);
	hipMemcpy(d_message, message, size_message, hipMemcpyHostToDevice);

	byte_sub_kernel <<<dim_grid, dim_block >>>(d_message);

	hipMemcpy(message, d_message, size_message, hipMemcpyDeviceToHost);
	hipFree(d_message);
}

// Inverse byte substitution (S-Boxes) can be parallel
void AES::byte_sub_inv(unsigned char *message)
{
	dim3 dim_block(KEY_BLOCK);
	dim3 dim_grid(1);

	int size_message = sizeof(message);
	unsigned char *d_message;
	hipMalloc((void **)&d_message, size_message);
	hipMemcpy(d_message, message, size_message, hipMemcpyHostToDevice);

	byte_sub_inv_kernel <<<dim_grid, dim_block>>>(d_message);

	hipMemcpy(message, d_message, size_message, hipMemcpyDeviceToHost);
	hipFree(d_message);
}

// Shift rows - can be parallel
// B0, B4, B8, B12 stays the same
void AES::shift_rows(unsigned char *message)
{
	dim3 dim_block(3);
	dim3 dim_grid(1);

	int size_message = sizeof(message);
	unsigned char *d_message;
	hipMalloc((void **)&d_message, size_message);
	hipMemcpy(d_message, message, size_message, hipMemcpyHostToDevice);

	shift_rows_kernel <<<dim_grid, dim_block>>>(d_message);

	hipMemcpy(message, d_message, size_message, hipMemcpyDeviceToHost);
	hipFree(d_message);
}

// Inverse shift rows - can be parallel
// C0, C4, C8, C12 stays the same
void AES::shift_rows_inv(unsigned char *message)
{
	dim3 dim_block(3);
	dim3 dim_grid(1);

	int size_message = sizeof(message);
	unsigned char *d_message;
	hipMalloc((void **)&d_message, size_message);
	hipMemcpy(d_message, message, size_message, hipMemcpyHostToDevice);

	shift_rows_inv_kernel<<<dim_grid, dim_block>>>(d_message);

	hipMemcpy(message, d_message, size_message, hipMemcpyDeviceToHost);
	hipFree(d_message);
}

// Mix column - can be parallel
void AES::mix_columns(unsigned char *message)
{
	dim3 dim_block(MIX_COLUMN_LIMIT);
	dim3 dim_grid(1);

	int size_message = sizeof(message);
	unsigned char *d_message;
	hipMalloc((void **)&d_message, size_message);
	hipMemcpy(d_message, message, size_message, hipMemcpyHostToDevice);

	mix_columns_kernel <<<dim_grid, dim_block>>>(d_message);

	hipMemcpy(message, d_message, size_message, hipMemcpyDeviceToHost);
	hipFree(d_message);
}

// Inverse mix column
void AES::mix_columns_inv(unsigned char *message)
{
	dim3 dim_block(MIX_COLUMN_LIMIT);
	dim3 dim_grid(1);

	int size_message = sizeof(message);
	unsigned char *d_message;
	hipMalloc((void **)&d_message, size_message);
	hipMemcpy(d_message, message, size_message, hipMemcpyHostToDevice);

	mix_columns_inv_kernel <<<dim_grid, dim_block>>>(d_message);

	hipMemcpy(message, d_message, size_message, hipMemcpyDeviceToHost);
	hipFree(d_message);
}

void AES::key_addition(unsigned char *message, const int &r)
{
	dim3 dim_block(KEY_BLOCK);
	dim3 dim_grid(1);

	int size_message = sizeof(message);
	unsigned char *d_message;
	hipMalloc((void **)&d_message, size_message);
	hipMemcpy(d_message, message, size_message, hipMemcpyHostToDevice);

	int size_subkey = sizeof(m_subkeys[r]);
	unsigned char *d_subkey = &m_subkeys[r][0];
	hipMalloc((void **)&d_subkey, size_subkey);
	hipMemcpy(d_subkey, &m_subkeys[r], size_subkey, hipMemcpyHostToDevice);

	key_addition_kernel <<<dim_grid, dim_block>>>(d_message, d_subkey);

	hipMemcpy(message, d_message, size_message, hipMemcpyDeviceToHost);
	hipFree(d_message);
	hipFree(d_subkey);
}